#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
* particleSystem_cuda.cu
*
* This file contains the functions that call the CUDA kernels. This is where you
* would need to add functions who call the kernels in any of the GPU code. Their
* definitions reside in particleSystem.cuh.
*
* If you have a function that require the specific particles group parameters then
* you need to send the d_gparams_array. With any array on the GPU you must send the
* pointer to that array that you allocated. This goes for the existing arrays as well.
* 
* Existing CUDA kernel function reside in particles_kernel_impl.cuh but new files have
* been added for position integration as well as the sphkernels. If you need new kernel
* files just use a .cuh file because for some reason visual studio has problems with
* function definitions being redefined. Put the implementation in the .cuh file, there
* doesnt need to be any declarations.
*
*/

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <helper_functions.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_impl.cuh"
#include "integration.cuh"

using namespace std;

extern "C"
{

    void cudaInit(int argc, char **argv)
    {
        int devID;

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        devID = findCudaDevice(argc, (const char **)argv);

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }

    void cudaGLInit(int argc, char **argv)
    {
        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        findCudaGLDevice(argc, (const char **)argv);
    }

    void allocateArray(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void threadSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

    void copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
    }

    void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsNone));
    }

    void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
    }

    void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
    {
        void *ptr;
        checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
        size_t num_bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
                                                             *cuda_vbo_resource));
        return ptr;
    }

    void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
    }

    void copyArrayFromDevice(void *host, const void *device,
                             struct hipGraphicsResource **cuda_vbo_resource, int size)
    {
        if (cuda_vbo_resource)
        {
            device = mapGLBufferObject(cuda_vbo_resource);
        }

        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

        if (cuda_vbo_resource)
        {
            unmapGLBufferObject(*cuda_vbo_resource);
        }
    }

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

	/**
	* void integrate()
	* 
	* Function that calls the position integration kernel specified by the init file. New
	* case statements should be added for the new kernel function which will be defined in
	* integration.cuh
	*
	* @param pos pointer to the position array in gpu memory
	* @param vel pointer to the velocity array in gpu memory 
	* @param deltaTime time step specified by the init file
	* @param numParticles the total number of particles in the simulation, ie array lengths
	* @param int_type enum which is defined in particles_kernel.cuh, specifies the integration method
	*/
	void integrate(float *pos,
		                 float *vel,
						 float deltaTime,
						 uint numParticles,
						 Type int_type)
	{
		uint numThreads, numBlocks;
		computeGridSize(numParticles, 256, numBlocks, numThreads);

		switch(int_type){
			case EULER:
				euler<<< numBlocks, numThreads >>>((float4 *) pos, 
											(float4 *) vel,
											deltaTime,
											numParticles);
				break;
			case VERLET:
				//Add CUDA kernel call just like one above
				break;
			default:
				break;
		}
		
	}

    void calcHash(uint  *gridParticleHash,
                  uint  *gridParticleIndex,
                  float *pos,
                  int    numParticles)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
                                               gridParticleIndex,
                                               (float4 *) pos,
                                               numParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     float *sortedPos,
                                     float *sortedVel,
                                     uint  *gridParticleHash,
                                     uint  *gridParticleIndex,
                                     float *oldPos,
                                     float *oldVel,
                                     uint   numParticles,
                                     uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

#if USE_TEX
        checkCudaErrors(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
#endif

        uint smemSize = sizeof(uint)*(numThreads+1);
        reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
            cellStart,
            cellEnd,
            (float4 *) sortedPos,
            (float4 *) sortedVel,
            gridParticleHash,
            gridParticleIndex,
            (float4 *) oldPos,
            (float4 *) oldVel,
            numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(oldVelTex));
#endif
    }

	/**
	* void collide()
	*
	* Main calculation function for collisions, this calls many different functions inside
	* the main kernel.
	*
	* @param newVel array containing the resulting calculated velocity where the vectors are at the original particles index
	* @param newForce array containing the resulting calculated force where the vectors are at the original particles index
	* @param sortedPos array containing the sorted particles where particles in close proximity are grouped together
	* @param sortedVel array containing the sorted particles where particles in close proximity are grouped together
	* @param gridParticleIndex array containing the particles new sorted index
	* @param cellStart array containing the index into the sorted arrays where each cell starts
	* @param cellEnd array containing the index into the sorted arrays where each cell ends
	* @param numParticles number of particles in the simulation
	* @param numCells number of cells that the 3d cube is divided into
	* @param d_gparams_array pointer to the group params array in gpu memory
	*/
    void collide(float *newVel,
				 float *newForce,
                 float *sortedPos,
                 float *sortedVel,
                 uint  *gridParticleIndex,
                 uint  *cellStart,
                 uint  *cellEnd,
                 uint   numParticles,
                 uint   numCells,
				 GroupParams* d_gparams_array)
    {
#if USE_TEX
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
#endif

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        collideD<<< numBlocks, numThreads >>>((float4 *)newVel,
											  (float4 *)newForce,
                                              (float4 *)sortedPos,
                                              (float4 *)sortedVel,
                                              gridParticleIndex,
                                              cellStart,
                                              cellEnd,
                                              numParticles,
											  d_gparams_array);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

#if USE_TEX
        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(oldVelTex));
        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
#endif
    }


    void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                            thrust::device_ptr<uint>(dGridParticleIndex));
    }

}   // extern "C"
